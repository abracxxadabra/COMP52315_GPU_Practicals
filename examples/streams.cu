
#include <hip/hip_runtime.h>
#include <iostream>

#define THREADS_PER_BLOCK 256
#define BLOCKS 10

__global__ void kernelA(int *a, int n) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadId; i < n; i += gridDim.x * blockDim.x) {
    a[i] = a[i] + 1;
  }
}

__global__ void kernelB(int *b, int n) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadId; i < n; i += gridDim.x * blockDim.x) {
    b[i] = b[i] * 2;
  }
}

int main() {
  long long N = 1000000;
  int *a, *b;
  hipMallocManaged(&a, N * sizeof(int));
  hipMallocManaged(&b, N * sizeof(int));

  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i;
  }

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  kernelA<<<BLOCKS,THREADS_PER_BLOCK, 0, stream1>>>(a, N);
  kernelB<<<BLOCKS, THREADS_PER_BLOCK, 0, stream2>>>(b, N);
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  for (int i = 0; i < 10; i++) {
    std::cout << a[i] << " ";
  }
  std::cout << std::endl;

  for (int i = 0; i < 10; i++) {
    std::cout << b[i] << " ";
  }
  std::cout << std::endl;

  hipFree(a);
  hipFree(b);

  EXIT_SUCCESS;
}
